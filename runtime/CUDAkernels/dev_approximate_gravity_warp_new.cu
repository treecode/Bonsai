#include "hip/hip_runtime.h"
#include "bonsai.h"
// #include "support_kernels.cu0
#include <stdio.h>
#include "../profiling/bonsai_timing.h"
PROF_MODULE(dev_approximate_gravity);


#include "node_specs.h"

#ifdef WIN32
#define M_PI        3.14159265358979323846264338328
#endif

#define WARP_SIZE2 5
#define WARP_SIZE  32

#if NCRIT > 4*WARP_SIZE
#error "NCRIT in include/node_specs.h must be <= WARP_SIZE"
#endif


#define laneId (threadIdx.x & (WARP_SIZE - 1))
#define warpId (threadIdx.x >> WARP_SIZE2)

#define BTEST(x) (-(int)(x))

#define FULL_MASK 0xffffffff


#if 1
#define _QUADRUPOLE_
#endif

/***********************************/
/***** DENSITY   ******************/

static __device__ __forceinline__ void computeDensityAndNgb(
    const float r2, const float hinv2, const float mass,
    float &density, float &nb)
{
#if 0  /* full kernel for reference */
  const float hinv = 1.0f/h;
  const float hinv2 = hinv*hinv;
  const float hinv3 = hinv*hinv2;
  const float C     = 3465.0f/(512.0f*M_PI)*hinv3;
  const float q2    = r2*hinv2;
  const float rho   = fmaxf(0.0f, 1.0f - q2);
  nb      += ceilf(rho);
  const float rho2 = rho*rho;
  density += C * rho2*rho2;
#else
  const float rho   = fmaxf(0.0f, 1.0f - r2*hinv2);   /* fma, fmax */
  const float rho2  = rho*rho;                        /* fmul */
  density += rho2*rho2;                               /* fma */
  nb      += ceilf(rho2);                             /* fadd, ceil */

  /*2x fma, 1x fmul, 1x fadd, 1x ceil, 1x fmax */
  /* total: 6 flops or 8 flops with ceil&fmax */
#endif
}

#if 0
static __device__ __forceinline__ float adjustH(const float h_old, const float nnb)
{
	const float nbDesired 	= 42;
	const float f      	= 0.5f * (1.0f + cbrtf(nbDesired / nnb));
	const float fScale 	= max(min(f, 1.2), 0.8);
	return (h_old*fScale);
}
#endif





/************************************/
/*********   PREFIX SUM   ***********/
/************************************/

static __device__ __forceinline__ uint shfl_scan_add_step(uint partial, uint up_offset)
{
    //"shfl.up.b32 r0|p, %1, %2, 0;"
  uint result;
  asm(
      "{.reg .u32 r0;"
      ".reg .pred p;"
      "shfl.sync.up.b32 r0|p, %1, %2, 0, 0xffffffff;"
      "@p add.u32 r0, r0, %3;"
      "mov.u32 %0, r0;}"
      : "=r"(result) : "r"(partial), "r"(up_offset), "r"(partial));
  return result;
}

static __device__ __forceinline__ int lanemask_lt()
{
  int mask;
  asm("mov.u32 %0, %lanemask_lt;" : "=r" (mask));
  return mask;
}

static __device__ __forceinline__ int lanemask_le()
{
  int mask;
  asm("mov.u32 %0, %lanemask_le;" : "=r" (mask));
  return mask;
}

static __device__ __forceinline__ int ShflSegScanStepB(
            int partial,
            uint distance,
            uint up_offset)
{
//      "shfl.up.b32 r0, %1, %2, 0;"
  asm(
      "{.reg .u32 r0;"
      ".reg .pred p;"
      "shfl.sync.up.b32 r0, %1, %2, 0, 0xffffffff;"
      "setp.le.u32 p, %2, %3;"
      "@p add.u32 %1, r0, %1;"
      "mov.u32 %0, %1;}"
      : "=r"(partial) : "r"(partial), "r"(up_offset), "r"(distance));
  return partial;
}

  template<const int SIZE2>
static __device__ __forceinline__ int inclusive_segscan_warp_step(int value, const int distance)
{
  for (int i = 0; i < SIZE2; i++)
    value = ShflSegScanStepB(value, distance, 1<<i);
  return value;
}

  template <const int levels>
static __device__ __forceinline__ uint inclusive_scan_warp(const int sum)
{
  uint mysum = sum;
#pragma unroll
  for(int i = 0; i < levels; ++i)
    mysum = shfl_scan_add_step(mysum, 1 << i);
  return mysum;
}

/*********************/

static __device__ __forceinline__ int2 warpIntExclusiveScan(const int value)
{
  const int sum = inclusive_scan_warp<WARP_SIZE2>(value);
  return make_int2(sum-value, __shfl_sync(FULL_MASK, sum, WARP_SIZE-1, WARP_SIZE));
}

static __device__ __forceinline__ int2 warpBinExclusiveScan(const bool p)
{
  const unsigned int b = __ballot_sync(FULL_MASK, p);
  return make_int2(__popc(b & lanemask_lt()), __popc(b));
}


static __device__ __forceinline__ int2 inclusive_segscan_warp(
    const int packed_value, const int carryValue)
{
  const int  flag = packed_value < 0;
  const int  mask = -flag;
  const int value = (~mask & packed_value) + (mask & (-1-packed_value));

  const int flags = __ballot_sync(FULL_MASK, flag);

  const int dist_block = __clz(__brev(flags));

  const int distance = __clz(flags & lanemask_le()) + laneId - 31;
  const int val = inclusive_segscan_warp_step<WARP_SIZE2>(value, min(distance, laneId));
  return make_int2(val + (carryValue & (-(laneId < dist_block))), __shfl_sync(FULL_MASK, val, WARP_SIZE-1, WARP_SIZE));
}

/**** binary scans ****/


#if 0
static __device__ int warp_exclusive_scan(const bool p, int &psum)
{
  const unsigned int b = __ballot(p);
  psum = __popc(b & lanemask_lt());
  return __popc(b);
}
static __device__ int warp_exclusive_scan(const bool p)
{
  const int b = __ballot(p);
  return __popc(b & lanemask_lt());
}
#endif


/**************************************/
/*************** Tree walk ************/
/**************************************/

  template<int SHIFT>
__forceinline__ static __device__ int ringAddr(const int i)
{
  return (i & ((CELL_LIST_MEM_PER_WARP<<SHIFT) - 1));
}


/*********** Forces *************/

static __device__ __forceinline__ float4 add_acc(
    float4 acc,  const float4 pos,
    const float massj, const float3 posj,
    const float eps2,
    float2 &density)
{
#if 1  // to test performance of a tree-walk
  const float3 dr = make_float3(posj.x - pos.x, posj.y - pos.y, posj.z - pos.z);

  const float r2     = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
  const float r2eps  = r2 + eps2;
  const float rinv   = rsqrtf(r2eps);
  const float rinv2  = rinv*rinv;
  const float mrinv  = massj * rinv;
  const float mrinv3 = mrinv * rinv2;

  acc.w -= mrinv;
  acc.x += mrinv3 * dr.x;
  acc.y += mrinv3 * dr.y;
  acc.z += mrinv3 * dr.z;

  computeDensityAndNgb(r2,pos.w,massj,density.x,density.y);
#endif

  return acc;
}
template<int NI, bool FULL>
static __device__ __forceinline__ void directAcc(
    float4 acc_i[NI],
    const float4 pos_i[NI],
    const int ptclIdx,
    const float eps2,
    float2 density_i[NI],
    const real4  *body_pos)
{
  const float4 M0 = (FULL || ptclIdx >= 0) ? body_pos[ptclIdx] : make_float4(0.0f, 0.0f, 0.0f, 0.0f);

//#pragma unroll
  for (int j = 0; j < WARP_SIZE; j++)
  {
    const float4 jM0 = make_float4(__shfl_sync(FULL_MASK, M0.x, j), __shfl_sync(FULL_MASK, M0.y, j),
                                   __shfl_sync(FULL_MASK, M0.z, j), __shfl_sync(FULL_MASK, M0.w,j));
    const float  jmass = jM0.w;
    const float3 jpos  = make_float3(jM0.x, jM0.y, jM0.z);
#pragma unroll
    for (int k = 0; k < NI; k++)
      acc_i[k] = add_acc(acc_i[k], pos_i[k], jmass, jpos, eps2, density_i[k]);
  }
}


#ifdef _QUADRUPOLE_

static __device__ __forceinline__ float4 add_acc(
    float4 acc,
    const float4 pos,
    const float mass, const float3 com,
    const float4 Q0,  const float4 Q1, float eps2,
    float2 &density)
{
#if 1
  const float3 dr = make_float3(pos.x - com.x, pos.y - com.y, pos.z - com.z);
  const float  r2 = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z + eps2;

  const float rinv  = rsqrtf(r2);
  const float rinv2 = rinv *rinv;
  const float mrinv  =  mass*rinv;
  const float mrinv3 = rinv2*mrinv;
  const float mrinv5 = rinv2*mrinv3;
  const float mrinv7 = rinv2*mrinv5;   // 16

  float  D0  =  mrinv;
  float  D1  = -mrinv3;
  float  D2  =  mrinv5*(  3.0f);
  float  D3  =  mrinv7*(-15.0f); // 3

  const float q11 = Q0.x;
  const float q22 = Q0.y;
  const float q33 = Q0.z;
  const float q12 = Q1.x;
  const float q13 = Q1.y;
  const float q23 = Q1.z;

  const float  q  = q11 + q22 + q33;
  const float3 qR = make_float3(
      q11*dr.x + q12*dr.y + q13*dr.z,
      q12*dr.x + q22*dr.y + q23*dr.z,
      q13*dr.x + q23*dr.y + q33*dr.z);
  const float qRR = qR.x*dr.x + qR.y*dr.y + qR.z*dr.z;  // 22

  acc.w  -= D0 + 0.5f*(D1*q + D2*qRR);
  float C = D1 + 0.5f*(D2*q + D3*qRR);
  acc.x  += C*dr.x + D2*qR.x;
  acc.y  += C*dr.y + D2*qR.y;
  acc.z  += C*dr.z + D2*qR.z;               // 23

// total: 16 + 3 + 22 + 23 = 64 flops

  return acc;
#endif
}

template<int NI, bool FULL>
static __device__ __forceinline__ void approxAcc(
    float4 acc_i[NI],
    const float4 pos_i[NI],
    float2 dens_i[NI],
    const int cellIdx,
    const float eps2,
    const real4 *multipole_data)
{
  const int cellAddr = cellIdx + cellIdx + cellIdx;
  float4 M0, Q0, Q1;
  if (FULL || cellIdx >= 0)
  {
    M0 = multipole_data[cellAddr];
    Q0 = multipole_data[cellAddr + 1];
    Q1 = multipole_data[cellAddr + 2];
  }
  else
    M0 = Q0 = Q1 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

  for (int j = 0; j < WARP_SIZE; j++)
  {
    const float4 jM0 = make_float4(__shfl_sync(FULL_MASK, M0.x, j), __shfl_sync(FULL_MASK, M0.y, j),
                                   __shfl_sync(FULL_MASK, M0.z, j), __shfl_sync(FULL_MASK, M0.w,j));
    const float4 jQ0 = make_float4(__shfl_sync(FULL_MASK, Q0.x, j), __shfl_sync(FULL_MASK, Q0.y, j),
                                   __shfl_sync(FULL_MASK, Q0.z, j), 0.0f);
    const float4 jQ1 = make_float4(__shfl_sync(FULL_MASK, Q1.x, j), __shfl_sync(FULL_MASK, Q1.y, j),
                                   __shfl_sync(FULL_MASK, Q1.z, j), 0.0f);
    const float  jmass = jM0.w;
    const float3 jpos  = make_float3(jM0.x, jM0.y, jM0.z);
#pragma unroll
      for (int k = 0; k < NI; k++)
        acc_i[k] = add_acc(acc_i[k], pos_i[k], jmass, jpos, jQ0, jQ1, eps2, dens_i[k]);

  }
}

#endif


/*******************************/
/****** Opening criterion ******/
/*******************************/

//Improved Barnes Hut criterium
static __device__ bool split_node_grav_impbh(
    const float4 nodeCOM,
    const float4 groupCenter,
    const float4 groupSize)
{
  //Compute the distance between the group and the cell
  float3 dr = make_float3(
      fabsf(groupCenter.x - nodeCOM.x) - (groupSize.x),
      fabsf(groupCenter.y - nodeCOM.y) - (groupSize.y),
      fabsf(groupCenter.z - nodeCOM.z) - (groupSize.z)
      );

  dr.x += fabsf(dr.x); dr.x *= 0.5f;
  dr.y += fabsf(dr.y); dr.y *= 0.5f;
  dr.z += fabsf(dr.z); dr.z *= 0.5f;

  //Distance squared, no need to do sqrt since opening criteria has been squared
  const float ds2    = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

  return (ds2 <= fabsf(nodeCOM.w));
}

//Minimum distance
__device__ bool split_node_grav_md(
    const float4 nodeCenter,
    const float4 nodeSize,
    const float4 groupCenter,
    const float4 groupSize)
{
  //Compute the distance between the group and the cell
  float3 dr = {fabs(groupCenter.x - nodeCenter.x) - (groupSize.x + nodeSize.x),
    fabs(groupCenter.y - nodeCenter.y) - (groupSize.y + nodeSize.y),
    fabs(groupCenter.z - nodeCenter.z) - (groupSize.z + nodeSize.z)};

  dr.x += fabs(dr.x); dr.x *= 0.5f;
  dr.y += fabs(dr.y); dr.y *= 0.5f;
  dr.z += fabs(dr.z); dr.z *= 0.5f;

  //Distance squared, no need to do sqrt since opening criteria has been squared
  float ds2    = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

  return (ds2 <= fabs(nodeCenter.w));
}




#define TEXTURES

template<int SHIFT, int BLOCKDIM2, int NI, bool INTCOUNT>
static __device__
uint2 approximate_gravity(
    float4 acc_i[NI],
    const float4 _pos_i[NI],
    const float4 groupPos,
    const float eps2,
    const uint2 top_cells,
    int *shmem,
    int *cellList,
    const float4 groupSize,
    float2 dens_i[NI],
    const real4 *body_pos,
    const real4 *multipole_data,
    const float4  *boxSizeInfo,
    const float4  *boxCenterInfo)
{
  const int laneIdx = threadIdx.x & (WARP_SIZE-1);

  /* this helps to unload register pressure */
  float4 pos_i[NI];
#pragma unroll 1
  for (int i = 0; i < NI; i++)
    pos_i[i] = _pos_i[i];

  uint2 interactionCounters = {0}; /* # of approximate and exact force evaluations */

#pragma unroll 1
  for (int i = 0; i < NI; i++)
    dens_i[i] = make_float2(0,0);


  volatile int *tmpList = shmem;

  int approxCellIdx, directPtclIdx;

  int directCounter = 0;
  int approxCounter = 0;

  for (int root_cell = top_cells.x; root_cell < top_cells.y; root_cell += WARP_SIZE)
    if (root_cell + laneIdx < top_cells.y)
      cellList[ringAddr<SHIFT>(root_cell - top_cells.x + laneIdx)] = root_cell + laneIdx;

  int nCells = top_cells.y - top_cells.x;

  int cellListBlock        = 0;
  int nextLevelCellCounter = 0;

  unsigned int cellListOffset = 0;

  /* process level with n_cells */
#if 1
  while (nCells > 0)
  {
    /* extract cell index from the current level cell list */
    const int cellListIdx = cellListBlock + laneIdx;
    const bool useCell    = cellListIdx < nCells;
    const int cellIdx     = cellList[ringAddr<SHIFT>(cellListOffset + cellListIdx)];
    cellListBlock += min(WARP_SIZE, nCells - cellListBlock);

    /* read from gmem cell's info */
    // Multiply with useCell  to prevent out of bound access, out of bound becomes idx '0'.
    // Previous texture lookup would allow out of bound look up as it would bound it to texture size.
    const float4 cellSize = boxSizeInfo[cellIdx*useCell];
    const float4 cellPos  = boxCenterInfo[cellIdx*useCell];
#if 1
    const float4 cellCOM = multipole_data[useCell*(cellIdx+cellIdx+cellIdx)];

    /* check if cell opening condition is satisfied */
    const float4 cellCOM1 = make_float4(cellCOM.x, cellCOM.y, cellCOM.z, cellPos.w);
    bool splitCell = split_node_grav_impbh(cellCOM1, groupPos, groupSize);
#else /*added by egaburov, see compute_propertiesD.cu for matching code */
    bool splitCell = split_node_grav_impbh(cellPos, groupPos, groupSize);
#endif

    /* compute first child, either a cell if node or a particle if leaf */
    const int cellData = __float_as_int(cellSize.w);
    const int firstChild =  cellData & 0x0FFFFFFF;
    const int nChildren  = (cellData & 0xF0000000) >> 28;

    if(cellData == 0xFFFFFFFF)
      splitCell = false;

    /**********************************************/
    /* split cells that satisfy opening condition */
    /**********************************************/

    const bool isNode = cellPos.w > 0.0f;

    {
      bool splitNode  = isNode && splitCell && useCell;

      /* use exclusive scan to compute scatter addresses for each of the child cells */
      const int2 childScatter = warpIntExclusiveScan(nChildren & (-splitNode));

      /* make sure we still have available stack space */
      if (childScatter.y + nCells - cellListBlock > (CELL_LIST_MEM_PER_WARP<<SHIFT))
        return make_uint2(0xFFFFFFFF,0xFFFFFFFF);

#if 1
      /* if so populate next level stack in gmem */
      if (splitNode)
      {
        const int scatterIdx = cellListOffset + nCells + nextLevelCellCounter + childScatter.x;
        for (int i = 0; i < nChildren; i++)
          cellList[ringAddr<SHIFT>(scatterIdx + i)] = firstChild + i;
      }
#else  /* use scan operation to accomplish steps above, doesn't bring performance benefit */
      int nChildren  = childScatter.y;
      int nProcessed = 0;
      int2 scanVal   = {0};
      const int offset = cellListOffset + nCells + nextLevelCellCounter;
      while (nChildren > 0)
      {
        tmpList[laneIdx] = 1;
        if (splitNode && (childScatter.x - nProcessed < WARP_SIZE))
        {
          splitNode = false;
          tmpList[childScatter.x - nProcessed] = -1-firstChild;
        }
        scanVal = inclusive_segscan_warp(tmpList[laneIdx], scanVal.y);
        if (laneIdx < nChildren)
          cellList[ringAddr<SHIFT>(offset + nProcessed + laneIdx)] = scanVal.x;
        nChildren  -= WARP_SIZE;
        nProcessed += WARP_SIZE;
      }
#endif
      nextLevelCellCounter += childScatter.y;  /* increment nextLevelCounter by total # of children */
    }

#if 1
    {
      /***********************************/
      /******       APPROX          ******/
      /***********************************/

      /* see which thread's cell can be used for approximate force calculation */
      const bool approxCell    = !splitCell && useCell;
      const int2 approxScatter = warpBinExclusiveScan(approxCell);

      /* store index of the cell */
      const int scatterIdx = approxCounter + approxScatter.x;
      tmpList[laneIdx] = approxCellIdx;
      if (approxCell && scatterIdx < WARP_SIZE)
        tmpList[scatterIdx] = cellIdx;

      approxCounter += approxScatter.y;

      /* compute approximate forces */
      if (approxCounter >= WARP_SIZE)
      {
        /* evalute cells stored in shmem */
        approxAcc<NI,true>(acc_i, pos_i, dens_i, tmpList[laneIdx], eps2, multipole_data);

        approxCounter -= WARP_SIZE;
        const int scatterIdx = approxCounter + approxScatter.x - approxScatter.y;
        if (approxCell && scatterIdx >= 0)
          tmpList[scatterIdx] = cellIdx;
        if (INTCOUNT)
          interactionCounters.x += WARP_SIZE*NI;
      }
      approxCellIdx = tmpList[laneIdx];
    }
#endif

#if 1
    {
      /***********************************/
      /******       DIRECT          ******/
      /***********************************/

      const bool isLeaf = !isNode;
      bool isDirect = splitCell && isLeaf && useCell;

      const int firstBody =   cellData & BODYMASK;
      const int     nBody = ((cellData & INVBMASK) >> LEAFBIT)+1;

      const int2 childScatter = warpIntExclusiveScan(nBody & (-isDirect));
      int nParticle  = childScatter.y;
      int nProcessed = 0;
      int2 scanVal   = {0};

      /* conduct segmented scan for all leaves that need to be expanded */
      while (nParticle > 0)
      {
        tmpList[laneIdx] = 1;
        if (isDirect && (childScatter.x - nProcessed < WARP_SIZE))
        {
          isDirect = false;
          tmpList[childScatter.x - nProcessed] = -1-firstBody;
        }
        scanVal = inclusive_segscan_warp(tmpList[laneIdx], scanVal.y);
        const int  ptclIdx = scanVal.x;

        if (nParticle >= WARP_SIZE)
        {
          directAcc<NI,true>(acc_i, pos_i, ptclIdx, eps2, dens_i, body_pos);
          nParticle  -= WARP_SIZE;
          nProcessed += WARP_SIZE;
          if (INTCOUNT)
            interactionCounters.y += WARP_SIZE*NI;
        }
        else
        {
          const int scatterIdx = directCounter + laneIdx;
          tmpList[laneIdx] = directPtclIdx;
          if (scatterIdx < WARP_SIZE)
            tmpList[scatterIdx] = ptclIdx;

          directCounter += nParticle;

          if (directCounter >= WARP_SIZE)
          {
            /* evalute cells stored in shmem */
            directAcc<NI,true>(acc_i, pos_i, tmpList[laneIdx], eps2, dens_i, body_pos);
            directCounter -= WARP_SIZE;
            const int scatterIdx = directCounter + laneIdx - nParticle;
            if (scatterIdx >= 0)
              tmpList[scatterIdx] = ptclIdx;
            if (INTCOUNT)
              interactionCounters.y += WARP_SIZE*NI;
          }
          directPtclIdx = tmpList[laneIdx];

          nParticle = 0;
        }
      }
    }
#endif

    /* if the current level is processed, schedule the next level */
    if (cellListBlock >= nCells)
    {
      cellListOffset += nCells;
      nCells = nextLevelCellCounter;
      cellListBlock = nextLevelCellCounter = 0;
    }

  }  /* level completed */
#endif

  if (approxCounter > 0)
  {
    approxAcc<NI,false>(acc_i, pos_i, dens_i, laneIdx < approxCounter ? approxCellIdx : -1, eps2, multipole_data);
    if (INTCOUNT)
      interactionCounters.x += approxCounter * NI;
    approxCounter = 0;
  }

  if (directCounter > 0)
  {
    directAcc<NI,false>(acc_i, pos_i, laneIdx < directCounter ? directPtclIdx : -1, eps2, dens_i, body_pos);
    if (INTCOUNT)
      interactionCounters.y += directCounter * NI;
    directCounter = 0;
  }

  return interactionCounters;
}

template<int SHIFT2, int BLOCKDIM2, bool ACCUMULATE>
static __device__
bool treewalk(
    const int bid,
    const float eps2,
    const uint2 node_begend,
    const int    *active_groups,
    const real4  *group_body_pos,
    const float4  *groupSizeInfo,
    const float4  *groupCenterInfo,
    int *shmem,
    int *lmem,
    float4 *acc_out,
    int2   *interactions,
    int    *ngb_out,
    int    *active_inout,
    float  *body_h,
    float2 *body_dens_out,
    const real4  *body_pos,
    const real4 *multipole_data,
    const float4  *boxSizeInfo,
    const float4  *boxCenterInfo)
{

  /*********** set necessary thread constants **********/
#ifdef DO_BLOCK_TIMESTEP
  real4 curGroupSize    = groupSizeInfo[active_groups[bid]];
#else
  real4 curGroupSize    = groupSizeInfo[bid];
#endif
  const int   groupData       = __float_as_int(curGroupSize.w);
  const uint body_addr        =   groupData & CRITMASK;
  const uint nb_i             = ((groupData & INVCMASK) >> CRITBIT) + 1;

#ifdef DO_BLOCK_TIMESTEP
  real4 group_pos       = groupCenterInfo[active_groups[bid]];
#else
  real4 group_pos       = groupCenterInfo[bid];
#endif


  uint body_i[2];
  const int ni = nb_i <= WARP_SIZE ? 1 : 2;
  body_i[0] = body_addr + laneId%nb_i;
  body_i[1] = body_addr + WARP_SIZE + laneId%(nb_i - WARP_SIZE);

  float4 pos_i[2];
  float4 acc_i[2];
  float2 dens_i[2];

  pos_i[0]   = group_body_pos[body_i[0]];
  pos_i[0].w = 1.0f/body_h[body_i[0]];
  pos_i[0].w *= pos_i[0].w;  /* .w stores 1/h^2 to speed up computations */
  if(ni > 1){       //Only read if we actually have ni == 2
    pos_i[1]   = group_body_pos[body_i[1]];
    pos_i[1].w = 1.0f/body_h[body_i[1]];
    pos_i[1].w *= pos_i[1].w;  /* .w stores 1/h^2 to speed up computations */
  }

  acc_i[0] = acc_i[1] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  dens_i[0] = dens_i[1] = make_float2(0.0f, 0.0f);


#if 0
  const bool INTCOUNT = false;
#else
  const bool INTCOUNT = true;
#endif
  uint2 counters = {0};
  {
    if (ni == 1)
      counters = approximate_gravity<SHIFT2, BLOCKDIM2, 1,INTCOUNT>(
          acc_i,
          pos_i,
          group_pos,
          eps2,
          node_begend,
          shmem,
          lmem,
          curGroupSize,
          dens_i,
          body_pos,
          multipole_data,
          boxSizeInfo,
          boxCenterInfo);
    else
      counters = approximate_gravity<SHIFT2, BLOCKDIM2, 2,INTCOUNT>(
          acc_i,
          pos_i,
          group_pos,
          eps2,
          node_begend,
          shmem,
          lmem,
          curGroupSize,
          dens_i,
          body_pos,
          multipole_data,
          boxSizeInfo,
          boxCenterInfo);
  }
  if(counters.x == 0xFFFFFFFF && counters.y == 0xFFFFFFFF)
    return false;

#if 0
  /* CUDA 8RC work around */
  if(bid < 0) // bid ==0 && laneId < nb_i && && threadIdx.x == 0)
  {
	  printf("TEST\n");
  	//printf("ON DEV [%d %d : %d %d ] ACC: %f %f %f %f INT: %d %d \n",
  	//		bid, threadIdx.x, nb_i, body_i[0],
  	//		acc_i[0].x,acc_i[0].y,acc_i[0].z,acc_i[0].w,
  	//		counters.x, counters.y);
  }
#endif

  if (laneId < nb_i)
  {
    const int addr = body_i[0];
    {
      const float hinv = 1.0f/body_h[addr];
      const float C   = 3465.0f/(512.0f*M_PI)*hinv*hinv*hinv;
      dens_i[0].x *= C;  /* scale rho */
    }
    if (ACCUMULATE)
    {
      acc_out     [addr].x += acc_i[0].x;
      acc_out     [addr].y += acc_i[0].y;
      acc_out     [addr].z += acc_i[0].z;
      acc_out     [addr].w += acc_i[0].w;


      body_dens_out[addr].x += dens_i[0].x;
      body_dens_out[addr].y += dens_i[0].y;
    }
    else
    {
      acc_out      [addr] =  acc_i[0];
      body_dens_out[addr] = dens_i[0];
    }
    //       ngb_out     [addr] = ngb_i;
    ngb_out     [addr] = addr; //JB Fixed this for demo
    active_inout[addr] = 1;
    if (ACCUMULATE)
    {
      interactions[addr].x += counters.x / ni;
      interactions[addr].y += counters.y / ni ;
    }
    else
    {
      interactions[addr].x = counters.x / ni;
      interactions[addr].y = counters.y / ni ;
    }
    if (ni == 2)
    {
      const int addr = body_i[1];
      {
        const float hinv = 1.0f/body_h[addr];
        const float C   = 3465.0f/(512.0f*M_PI)*hinv*hinv*hinv;
        dens_i[1].x *= C;  /* scale rho */
      }
      if (ACCUMULATE)
      {
        acc_out     [addr].x += acc_i[1].x;
        acc_out     [addr].y += acc_i[1].y;
        acc_out     [addr].z += acc_i[1].z;
        acc_out     [addr].w += acc_i[1].w;

        body_dens_out[addr].x += dens_i[1].x;
      	body_dens_out[addr].y += dens_i[1].y;
      }
      else
      {
        acc_out      [addr] =  acc_i[1];
        body_dens_out[addr] = dens_i[1];

//	body_h[addr] = adjustH(body_h[addr], dens_i[1].y);
      }

      //         ngb_out     [addr] = ngb_i;
      ngb_out     [addr] = addr; //JB Fixed this for demo
      active_inout[addr] = 1;
      if (ACCUMULATE)
      {
        interactions[addr].x += counters.x / ni;
        interactions[addr].y += counters.y / ni;
      }
      else
      {
        interactions[addr].x = counters.x / ni;
        interactions[addr].y = counters.y / ni;
      }
    }
  }

  return true;
}

template<bool ACCUMULATE, int BLOCKDIM2>
static __device__
void approximate_gravity_main(
    const int n_active_groups,
    int    n_bodies,
    float eps2,
    uint2 node_begend,
    int    *active_groups,
    real4  *body_pos,
    real4  *multipole_data,
    float4 *acc_out,
    real4  *group_body_pos,           //This can be different from body_pos
    int    *ngb_out,
    int    *active_inout,
    int2   *interactions,
    float4  *boxSizeInfo,
    float4  *groupSizeInfo,
    float4  *boxCenterInfo,
    float4  *groupCenterInfo,
    real4   *body_vel,
    int     *MEM_BUF,
    float   *body_h,
    float2  *body_dens)
{
  const int blockDim2 = BLOCKDIM2;
  const int shMemSize = 1 * (1 << blockDim2);
  __shared__ int shmem_pool[shMemSize];

  const int nWarps2 = blockDim2 - WARP_SIZE2;

  const int sh_offs = (shMemSize >> nWarps2) * warpId;
  int *shmem = shmem_pool + sh_offs;
  volatile int *shmemv = shmem;


#if 0
#define SHMODE
#endif

#ifdef SHMODE
  const int nWarps  = 1<<nWarps2;
  const int MAXFAILED = 64;
  __shared__ int failedList[MAXFAILED];
  __shared__ unsigned int failed;

  if (threadIdx.x == 0)
    failed = 0;
#endif

  __syncthreads();

  /*********** check if this block is linked to a leaf **********/

  int  bid  = gridDim.x * blockIdx.y + blockIdx.x;

  while(true)
  {
    if(laneId == 0)
    {
      bid         = atomicAdd(&active_inout[n_bodies], 1);
      shmemv[0]    = bid;
    }

    bid   = shmemv[0];

    if (bid >= n_active_groups) return;

    int *lmem = &MEM_BUF[(CELL_LIST_MEM_PER_WARP<<nWarps2)*blockIdx.x + CELL_LIST_MEM_PER_WARP*warpId];
    const bool success = treewalk<0,blockDim2,ACCUMULATE>(
        bid,
        eps2,
        node_begend,
        active_groups,
        group_body_pos,
        groupSizeInfo,
        groupCenterInfo,
        shmem,
        lmem,
        acc_out,
        interactions,
        ngb_out,
        active_inout,
        body_h,
        body_dens,
        body_pos,
        multipole_data,
        boxSizeInfo,
        boxCenterInfo);

#if 0
    if (bid % 10 == 0)
      success = false;
#endif

#ifdef SHMODE
    if (!success)
      if (laneId == 0)
        failedList[atomicAdd(&failed,1)] = bid;

    if (failed + nWarps >= MAXFAILED)
    {
      __syncthreads();
      if (warpId == 0)
      {
        int *lmem1 = &MEM_BUF[(CELL_LIST_MEM_PER_WARP<<nWarps2)*blockIdx.x];
        const int n = failed;
        failed = 0;
        for (int it = 0; it < n; it++)
        {
          const bool success = treewalk<nWarp2,blockDim2,ACCUMULATE>(
              failedList[it],
              eps2,
              node_begend,
              active_groups,
              group_body_pos,
              groupSizeInfo,
              groupCenterInfo,
              shmem,
              lmem1,
              acc_out,
              interactions,
              ngb_out,
              active_inout);
          assert(success);
        }
      }
      __syncthreads();
    }

#else

    //Try to get access to the big stack, only one block per time is allowed
    if (!success)
    {
      if(laneId == 0)
      {
        int res = atomicExch(&active_inout[n_bodies+1], 1); //If the old value (res) is 0 we can go otherwise sleep
        int waitCounter  = 0;
        while(res != 0)
        {
          //Sleep
          for(int i=0; i < (1024); i++)
            waitCounter += 1;

          //Test again
          shmem[0] = waitCounter;
          res = atomicExch(&active_inout[n_bodies+1], 1);
        }
      }

      int *lmem1 = &MEM_BUF[gridDim.x*(CELL_LIST_MEM_PER_WARP<<nWarps2)];
      const bool success = treewalk<8,blockDim2,ACCUMULATE>(
          bid,
          eps2,
          node_begend,
          active_groups,
          group_body_pos,
          groupSizeInfo,
          groupCenterInfo,
          shmem,
          lmem1,
          acc_out,
          interactions,
          ngb_out,
          active_inout,
          body_h,
          body_dens,
          body_pos,
          multipole_data,
          boxSizeInfo,
          boxCenterInfo);
      assert(success);

      if(laneId == 0)
        atomicExch(&active_inout[n_bodies+1], 0); //Release the lock
    }
#endif /* SHMODE */
  }     //end while
#undef SHMODE
}


  extern "C"
__launch_bounds__(NTHREAD,1024/NTHREAD)
  __global__ void
  dev_approximate_gravity(
      const int n_active_groups,
      int    n_bodies,
      float eps2,
      uint2 node_begend,
      int    *active_groups,
      real4  *body_pos,
      real4  *multipole_data,
      float4 *acc_out,
      real4  *group_body_pos,           //This can be different from body_pos
      int    *ngb_out,
      int    *active_inout,
      int2   *interactions,
      float4  *boxSizeInfo,
      float4  *groupSizeInfo,
      float4  *boxCenterInfo,
      float4  *groupCenterInfo,
      real4   *body_vel,
      int     *MEM_BUF,
      float   *body_h,
      float2  *body_dens)
{
  approximate_gravity_main<false, NTHREAD2>(
      n_active_groups,
      n_bodies,
      eps2,
      node_begend,
      active_groups,
      body_pos,
      multipole_data,
      acc_out,
      group_body_pos,           //This can be different from body_pos
      ngb_out,
      active_inout,
      interactions,
      boxSizeInfo,
      groupSizeInfo,
      boxCenterInfo,
      groupCenterInfo,
      body_vel,
      MEM_BUF,
      body_h,
      body_dens);
}


  extern "C"
__launch_bounds__(NTHREAD,1024/NTHREAD)
  __global__ void
  dev_approximate_gravity_let(
      const int n_active_groups,
      int    n_bodies,
      float eps2,
      uint2 node_begend,
      int    *active_groups,
      real4  *body_pos,
      real4  *multipole_data,
      float4 *acc_out,
      real4  *group_body_pos,           //This can be different from body_pos
      int    *ngb_out,
      int    *active_inout,
      int2   *interactions,
      float4  *boxSizeInfo,
      float4  *groupSizeInfo,
      float4  *boxCenterInfo,
      float4  *groupCenterInfo,
      real4   *body_vel,
      int     *MEM_BUF,
      float   *body_h,
      float2  *body_dens)
{
  approximate_gravity_main<true, NTHREAD2>(
      n_active_groups,
      n_bodies,
      eps2,
      node_begend,
      active_groups,
      body_pos,
      multipole_data,
      acc_out,
      group_body_pos,           //This can be different from body_pos
      ngb_out,
      active_inout,
      interactions,
      boxSizeInfo,
      groupSizeInfo,
      boxCenterInfo,
      groupCenterInfo,
      body_vel,
      MEM_BUF,
      body_h,
      body_dens);
}

